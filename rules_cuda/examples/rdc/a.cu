#include "hip/hip_runtime.h"
#include "b.cuh"
#include <stdio.h>

#define CUDA_CHECK(expr)                                                \
  do {                                                                  \
    hipError_t err = (expr);                                           \
    if (err != hipSuccess) {                                           \
      fprintf(stderr, "CUDA Error Code  : %d\n     Error String: %s\n", \
              err, hipGetErrorString(err));                            \
      exit(err);                                                        \
    }                                                                   \
  } while (0)

__global__ void foo() {
  __shared__ int a[N];
  a[threadIdx.x] = threadIdx.x;
  __syncthreads();

  g[threadIdx.x] = a[blockDim.x - threadIdx.x - 1];
  bar();
}

int main(void) {
  unsigned int i;
  int *dg, hg[N];
  int sum = 0;

  foo<<<1, N>>>();
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipGetSymbolAddress((void**)&dg, g));
  CUDA_CHECK(hipMemcpy(hg, dg, N * sizeof(int), hipMemcpyDeviceToHost));

  for (i = 0; i < N; i++) {
    sum += hg[i];
  }
  if (sum == 36) {
    printf("PASSED\n");
  } else {
    printf("FAILED (%d)\n", sum);
  }

  return 0;
}
