#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void kernel() {
  printf("cuda enabled\n");
}

void launch() {
  kernel<<<1, 1>>>();
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
}
